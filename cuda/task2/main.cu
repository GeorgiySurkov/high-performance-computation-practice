#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel для суммирования элементов массива
__global__ void sumArray(int *array, int *result, int n) {
    extern __shared__ int shared_data[];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;

    if (index < n) {
        shared_data[tid] = array[index];
    } else {
        shared_data[tid] = 0;
    }

    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2 * stride) == 0) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(result, shared_data[0]);
    }
}

void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        std::cerr << msg << ": " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void runSumArray(int n) {
    int *h_array, *d_array, *d_result;
    int h_result = 0;

    // Измерение времени инициализации и копирования данных
    hipEvent_t init_start, init_stop;
    hipEventCreate(&init_start);
    hipEventCreate(&init_stop);
    hipEventRecord(init_start);

    h_array = (int *)malloc(n * sizeof(int));
    hipMalloc(&d_array, n * sizeof(int));
    hipMalloc(&d_result, sizeof(int));

    // Инициализация массива случайными значениями
    for (int i = 0; i < n; ++i) {
        h_array[i] = rand() % 100;
    }

    hipMemcpy(d_array, h_array, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &h_result, sizeof(int), hipMemcpyHostToDevice);

    hipEventRecord(init_stop);
    hipEventSynchronize(init_stop);
    float init_time = 0;
    hipEventElapsedTime(&init_time, init_start, init_stop);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Измерение времени выполнения CUDA kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    sumArray<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>(d_array, d_result, n);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float kernel_time = 0;
    hipEventElapsedTime(&kernel_time, start, stop);

    hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Sum of array with " << n << " elements: " << h_result << std::endl;
    std::cout << "Initialization and memory copy time: " << init_time << " ms" << std::endl;
    std::cout << "Kernel execution time: " << kernel_time << " ms" << std::endl;

    free(h_array);
    hipFree(d_array);
    hipFree(d_result);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipEventDestroy(init_start);
    hipEventDestroy(init_stop);
}

int main() {
    int sizes[] = {10, 1000, 10000000};

    for (int size : sizes) {
        runSumArray(size);
    }

    return 0;
}
